#include "hip/hip_runtime.h"
#include <cstdio>
#include "hip/hip_runtime.h"
#include ""
#include "Plane.h"
#include <SFML\Graphics.hpp>


#define NUM_THREAD 1024

hipError_t computeMandelbrot(int, int, ComplexMandelbrot*, int*);

__global__ void mandelbrotKernel(int size, int iterations, ComplexMandelbrot* c_initial, ComplexMandelbrot* z_current, int* screen)
{
    int i = threadIdx.x + blockIdx.x*blockDim.x;
	if (i < size) {
		ComplexMandelbrot z = z_current[i];
		DATA_TYPE x, y;
		int k;
		for (k = 0; k <iterations; k++) {
			x = (z.real * z.real - z.imaginary * z.imaginary) + c_initial[i].real;
			y = (z.imaginary * z.real + z.real * z.imaginary) + c_initial[i].imaginary;

			if ((x * x + y * y) > 4.0) break;
			z.real = x;
			z.imaginary = y;
		}
		screen[i] += k;
	}
}

__global__ void resetScreenKernel(int size, int* screen) {
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if (i < size) {
		screen[i] = 0;
	}
}

__global__ void shaderScreenKernel(int size, int iterations, int* screen) {
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if (i < size) {
		if (screen[i] == iterations) screen[i] = 0;
		float t, r, g, b;
		if (screen[i] != 0) {
			//screen[i] = (Color)( 0xFF0000FF + (complexNumbers[i].counter << 13));
			t = (float)screen[i] / (float)iterations;
			r = 9 * (1 - t)*t*t*t;
			g = 15 * (1 - t)*(1 - t)*t*t;
			b = 8.5*(1 - t)*(1 - t)*(1 - t)*t;
			screen[i] = (static_cast<int>(r * 255)) + (static_cast<int>(g * 255) << 8) + (static_cast<int>(b * 255) << 16) + (static_cast<int>(255) << 24);
		}
	}
}


constexpr DATA_TYPE ZOOM_FACTOR = 1.3;

int main()
{
	DATA_TYPE MOVE_FACTOR = 0.2;

	ofstream parametersFile;
	parametersFile.open("Parameters.txt");
	if (!parametersFile.is_open()) {
		cout << "Couldn't open Paramters.txt\n";
	}

	Vue view(ASPECT_RATIO);
	/*
	view.xmin = -0.759130752537225395;
	view.xmax = -0.759130739373624519;
	view.ymin = -0.070381827302726005;
	view.ymax = -0.070381819903563639;

	MOVE_FACTOR = 0.000000000739984117;
	*/

	Plane plane(view);

	Event evnt;
	while (plane.isOpen()) {
		while (plane.pollEvent(evnt)) {
			if (evnt.type == Event::Closed) {
				plane.close();
			}
		}
		if (Keyboard::isKeyPressed(Keyboard::Z)) {
			view.move(0.0f, MOVE_FACTOR);
			plane.setWindow(view);
		}
		if (Keyboard::isKeyPressed(Keyboard::S)) {
			view.move(0.0f, -MOVE_FACTOR);
			plane.setWindow(view);
		}
		if (Keyboard::isKeyPressed(Keyboard::Q)) {
			view.move(ASPECT_RATIO * (-MOVE_FACTOR), 0.0f);
			plane.setWindow(view);
		}
		if (Keyboard::isKeyPressed(Keyboard::D)) {
			view.move(ASPECT_RATIO * MOVE_FACTOR, 0.0f);
			plane.setWindow(view);
		}
		if (Keyboard::isKeyPressed(Keyboard::E)) {
			view.zoom(ZOOM_FACTOR);
			plane.setWindow(view);
			MOVE_FACTOR /= ZOOM_FACTOR;
		}
		if (Keyboard::isKeyPressed(Keyboard::A)) {
			view.zoom(1.0f / ZOOM_FACTOR);
			plane.setWindow(view);
			MOVE_FACTOR *= ZOOM_FACTOR;
		}
		if (Keyboard::isKeyPressed(Keyboard::Up)) {
			int ITERATIONS = plane.getNumIterations();
			ITERATIONS < 100 ? ITERATIONS += 1 : ITERATIONS *= 1.3;
			cout << "ITERATIONS : " << ITERATIONS << "\n";
			plane.setNumIterations(ITERATIONS);
		}
		if (Keyboard::isKeyPressed(Keyboard::Down)) {
			int ITERATIONS = plane.getNumIterations();
			ITERATIONS < 100 ? ITERATIONS -= 1 : ITERATIONS /= 1.3;
			cout << "ITERATIONS : " << ITERATIONS << "\n";
			plane.setNumIterations(ITERATIONS);
		}
		if (Keyboard::isKeyPressed(Keyboard::R)) {
			plane.setNumIterations(1500);
			cout << "ITERATIONS : " << 1500 << "\n";
		}
		if (Keyboard::isKeyPressed(Keyboard::M)) {
			plane.setNumIterations(100000);
			cout << "ITERATIONS : " << 100000 << "\n";
		}
		if (Keyboard::isKeyPressed(Keyboard::O)) {
			plane.setNumIterations(1000000);
			cout << "ITERATIONS : " << 1000000 << "\n";
		}
		if (Mouse::isButtonPressed(Mouse::Button::Left)) {
			printf("Parameters :\n\
				Xmin = %.18lf,\n\
				Xmax = %.18lf,\n\
				Ymin = %.18lf,\n\
				Ymax = %.18lf\n\n\
				Mouse Cursor Position : %.18lf , %.18lf\n\n\
				Move factor : %.18lf\n\n\n",
				view.xmin,
				view.xmax,
				view.ymin,
				view.ymax,
				(DATA_TYPE)Mouse::getPosition().x * (view.xmax - view.xmin) / (DATA_TYPE)WIDTH,
				(DATA_TYPE)Mouse::getPosition().y * (view.ymax - view.ymin) / (DATA_TYPE)HEIGHT,
				MOVE_FACTOR);

			parametersFile << fixed << setprecision(numeric_limits<DATA_TYPE>::digits10 + 1) << "Parameters :\nview.xmin = " << view.xmin <<
				"\nview.xmax = " << view.xmax << "\nview.ymin = " << view.ymin << "\nview.ymax = " << view.ymax << "\n\nMouse Cursor : " <<
				(DATA_TYPE)Mouse::getPosition().x * (view.xmax - view.xmin) / (DATA_TYPE)WIDTH << " , " <<
				(DATA_TYPE)Mouse::getPosition().y * (view.ymax - view.ymin) / (DATA_TYPE)HEIGHT << "\n\n" <<
				"MOVE_FACTOR = " << MOVE_FACTOR << "\n\n\n";
		}

		plane.clear();
		computeMandelbrot(NUM_PIXEL, plane.getNumIterations(), plane.getComplexNumbers(), plane.getScreenPointer());
		plane.drawPlane();
		plane.display();
	}
	return 0;
}






// Helper function for using CUDA to add vectors in parallel.
hipError_t computeMandelbrot(int size, int iterations, ComplexMandelbrot* c, int* screen)
{
    ComplexMandelbrot *dev_c;
	ComplexMandelbrot *dev_z;
	int* dev_screen;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(ComplexMandelbrot));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

	cudaStatus = hipMalloc((void**)&dev_z, size * sizeof(ComplexMandelbrot));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_screen, size * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_c, c, size * sizeof(ComplexMandelbrot), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

	cudaStatus = hipMemcpy(dev_z, dev_c, size * sizeof(ComplexMandelbrot), hipMemcpyDeviceToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	// Reset dev_screen
	resetScreenKernel <<<(size + NUM_THREAD - 1) / NUM_THREAD, NUM_THREAD >>>(size, dev_screen);

	unsigned long long maxIterationsPerKernel = 10000/10;
	unsigned long long kernelCalls = iterations / maxIterationsPerKernel;

	for (int i = 0; i < kernelCalls; i++) {
		mandelbrotKernel <<<(size + NUM_THREAD - 1) / NUM_THREAD, NUM_THREAD >>>(size, maxIterationsPerKernel, dev_c, dev_z, dev_screen);
		// Check for any errors launching the kernel
		cudaStatus = hipGetLastError();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "mandelbrotKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
			goto Error;
		}
		// hipDeviceSynchronize waits for the kernel to finish, and returns
		// any errors encountered during the launch.
		cudaStatus = hipDeviceSynchronize();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching mandelbrotKernel!\n", cudaStatus);
			goto Error;
		}
	}

	mandelbrotKernel <<<(size + NUM_THREAD - 1) / NUM_THREAD, NUM_THREAD >>>(size, iterations % maxIterationsPerKernel, dev_c, dev_z, dev_screen);
	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "mandelbrotKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}
	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching mandelbrotKernel!\n", cudaStatus);
		goto Error;
	}


	// Apply shader to dev_screen
	shaderScreenKernel << <(size + NUM_THREAD - 1) / NUM_THREAD, NUM_THREAD >> >(size, iterations, dev_screen);

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(screen, dev_screen, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
	hipFree(dev_z);
	hipFree(dev_screen);
    
    return cudaStatus;
}
